#include<unistd.h>
#include<stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include<hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>

struct Point {
    float x;
    float y;
};
typedef struct Point Point;

// LOCAL
__global__ void data_kernel(Point* p_data, size_t numPoints, int seed,
    float leftX, float lowerY, float rightX, float upperY) {

    hiprandState_t state; // TODO : better setting of seed
    hiprand_init (seed + (128 * blockIdx.x) + (60000 * threadIdx.x), 0, 0, &state);
    float sizeX = rightX - leftX;
    float sizeY = upperY - lowerY;

    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    float x, y;
    while (index < numPoints) {
        x = hiprand_uniform (&state);
        y = hiprand_uniform (&state);
        p_data[index].x = (x * sizeX) + leftX + 0.1;
        p_data[index].y = (y * sizeY) + lowerY + 0.1;
        index += blockDim.x * gridDim.x;
    }
}


// Prevent name mangling
extern "C" {
    void c_cudaAlloc(Point ** points, size_t numPoints);

    void c_callKernel(int numBlocks, int threadsCount, Point * p_data, size_t numPoints, int seed,
        float leftX, float lowerY, float rightX, float upperY);

    void setCudaDeviceByRank(int myrank);

    void freeCudaMemory(Point * pts);
}

void c_cudaAlloc(Point ** points, size_t numPoints) {
    hipMallocManaged(points, numPoints * sizeof(Point));
}

void setCudaDeviceByRank(int myrank) {
    hipError_t cE = hipSuccess;
    int cudaDeviceCount = -1;
    int assignedCudaDevice = -1;

    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
        printf(" Unable to determine cuda device count, error is %d, count is %d\n",
            cE, cudaDeviceCount );
        exit(-1);
    }
    if( (cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess )
    {
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
            myrank, (myrank % cudaDeviceCount), cE);
        exit(-1);
    }

    if( (cE = hipGetDevice( &assignedCudaDevice )) != hipSuccess )
    {
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n", 
            myrank, (myrank % cudaDeviceCount), cE);
        exit(-1);
    }

    if( assignedCudaDevice != (myrank % cudaDeviceCount) )
    {
        printf("MPI Rank %d: assignedCudaDevice %d NOT EQ to (myrank(%d) mod cudaDeviceCount(%d)) \n",
            myrank, assignedCudaDevice, myrank, cudaDeviceCount );
        exit(-1);
    }
    printf("Rank %d: My Cuda Device is %d\n", myrank, assignedCudaDevice);
}

void freeCudaMemory(Point * pts) {
    hipFree(pts);
}

void c_callKernel(int numBlocks, int threadsCount, Point * p_data, size_t numPoints, int seed,
        float leftX, float lowerY, float rightX, float upperY) {
    data_kernel<<<numBlocks, threadsCount>>>(p_data, numPoints, seed,
        leftX, lowerY, rightX, upperY);
    hipDeviceSynchronize();
}
