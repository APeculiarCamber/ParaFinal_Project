#include<unistd.h>
#include<stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include<hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>

struct Point {
    float x;
    float y;
};
typedef struct Point Point;

// LOCAL
__global__ void data_kernel(Point* p_data, size_t numPoints, int seed,
    float leftX, float lowerY, float rightX, float upperY) {

    hiprandState_t state; // TODO : better setting of seed
    hiprand_init (seed + (128 * blockIdx.x) + (60000 * threadIdx.x), 0, 0, &state);
    float sizeX = rightX - leftX;
    float sizeY = upperY - lowerY;

    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    float x, y;
    while (index < numPoints) {
        x = hiprand_uniform (&state);
        y = hiprand_uniform (&state);
        p_data[index].x = (x * sizeX) + leftX;
        p_data[index].y = (y * sizeY) + lowerY;
        index += blockDim.x * gridDim.x;
    }
}


// Prevent name mangling
extern "C" {
    void c_cudaAlloc(Point ** points, size_t numPoints);

    void c_callKernel(int numBlocks, int threadsCount, Point * p_data, size_t numPoints, int seed,
        float leftX, float lowerY, float rightX, float upperY);
}

void c_cudaAlloc(Point ** points, size_t numPoints) {
    hipMallocManaged(points, numPoints * sizeof(Point));
}

void c_callKernel(int numBlocks, int threadsCount, Point * p_data, size_t numPoints, int seed,
        float leftX, float lowerY, float rightX, float upperY) {
    data_kernel<<<numBlocks, threadsCount>>>(p_data, numPoints, seed,
        leftX, lowerY, rightX, upperY);
    hipDeviceSynchronize();
}