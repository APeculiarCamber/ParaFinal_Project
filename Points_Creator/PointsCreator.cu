#include<unistd.h>
#include<stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include<hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>

#define PI 3.14159265359

struct Point {
    float x;
    float y;
};
typedef struct Point Point;

// LOCAL
__global__ void data_kernel(Point* p_data, size_t numPoints, int seed,
    float leftX, float lowerY, float rightX, float upperY) {

    hiprandState_t state;
    hiprand_init (seed + (128 * blockIdx.x) + (60000 * threadIdx.x), 0, 0, &state);
    float sizeX = rightX - leftX;
    float sizeY = upperY - lowerY;

    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    float x, y;
    while (index < numPoints) {
        x = hiprand_uniform (&state);
        y = hiprand_uniform (&state);
        p_data[index].x = (x * sizeX) + leftX;
        p_data[index].y = (y * sizeY) + lowerY;
        index += blockDim.x * gridDim.x;
    }
}

// LOCAL
__global__ void data_circle_kernel(Point* p_data, size_t numPoints, int seed,
    float radius) {

    hiprandState_t state;
    hiprand_init (seed + (128 * blockIdx.x) + (60000 * threadIdx.x), 0, 0, &state);

    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    while (index < numPoints) {
        float r = radius * sqrt(hiprand_uniform (&state));
        float t = hiprand_uniform (&state) * 2 * PI;

        p_data[index].x = radius + (r * cos(t));
        p_data[index].y = radius + (r * sin(t));
        index += blockDim.x * gridDim.x;
    }
}


// Prevent name mangling
extern "C" {
    void c_cudaAlloc(Point ** points, size_t numPoints);

    void c_callKernel(int numBlocks, int threadsCount, Point * p_data, size_t numPoints, int seed,
        float leftX, float lowerY, float rightX, float upperY);

    void c_callCircleKernel(int numBlocks, int threadsCount, Point * p_data, 
        size_t numPoints, int seed, float radius);

    void setCudaDeviceByRank(int myrank);

    void freeCudaMemory(Point * pts);
}

void c_cudaAlloc(Point ** points, size_t numPoints) {
    hipMallocManaged(points, numPoints * sizeof(Point));
}

void setCudaDeviceByRank(int myrank) {
    hipError_t cE = hipSuccess;
    int cudaDeviceCount = -1;
    int assignedCudaDevice = -1;

    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
        printf(" Unable to determine cuda device count, error is %d, count is %d\n",
            cE, cudaDeviceCount );
        exit(-1);
    }
    if( (cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess )
    {
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
            myrank, (myrank % cudaDeviceCount), cE);
        exit(-1);
    }

    if( (cE = hipGetDevice( &assignedCudaDevice )) != hipSuccess )
    {
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n", 
            myrank, (myrank % cudaDeviceCount), cE);
        exit(-1);
    }

    if( assignedCudaDevice != (myrank % cudaDeviceCount) )
    {
        printf("MPI Rank %d: assignedCudaDevice %d NOT EQ to (myrank(%d) mod cudaDeviceCount(%d)) \n",
            myrank, assignedCudaDevice, myrank, cudaDeviceCount );
        exit(-1);
    }
    printf("Rank %d: My Cuda Device is %d\n", myrank, assignedCudaDevice);
}

void freeCudaMemory(Point * pts) {
    hipFree(pts);
}

void c_callKernel(int numBlocks, int threadsCount, Point * p_data, size_t numPoints, int seed,
        float leftX, float lowerY, float rightX, float upperY) {
    data_kernel<<<numBlocks, threadsCount>>>(p_data, numPoints, seed,
        leftX, lowerY, rightX, upperY);
    hipDeviceSynchronize();
}

void c_callCircleKernel(int numBlocks, int threadsCount, Point * p_data, size_t numPoints, int seed,
        float radius) {
    data_circle_kernel<<<numBlocks, threadsCount>>>(p_data, numPoints, seed,
        radius);
    hipDeviceSynchronize();
}
