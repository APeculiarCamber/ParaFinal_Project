#include<unistd.h>
#include<stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include<hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>

struct Point {
    float x;
    float y;
};

__global__ void data_kernel(Point* p_data, size_t numPoints,
    float leftX, float upperY, float rightX, float lowerY) {

    hiprandState_t state;
    hiprand_init (0x2340238, 0, 0, &state);
    float sizeX = rightX - leftX;
    float sizeY = upperY - lowerY;

    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    float x, y;
    while (index < numPoints) {
        x = hiprand_uniform (&state);
        y = hiprand_uniform (&state);
        p_data[index].x = (x * sizeX) + leftX;
        p_data[index].y = (y * sizeY) + lowerY;
        index += blockDim.x * gridDim.x;
    }
}


// TODO : add MPI and parallel file processing for even larger files
int main(int argc, char* argv[])
{
    if (argc < 7) {
        printf("FORMAT: %s <num points> <num threads> <left x> <upper y> <right x> <lower y>", argv[0]);
        return false;
    }
    size_t numPoints = 0;
    if (1 != sscanf(argv[1], "%zu", &numPoints))
        return false;
    int threadsCount = atoi(argv[2]);
    float leftX = atof(argv[3]);
    float upperY = atof(argv[4]);
    float rightX = atof(argv[5]);
    float lowerY = atof(argv[6]);
    printf("Number elements is %zu\n", numPoints);
    
    Point * p_data;
    hipMallocManaged((void**)&p_data, numPoints * sizeof(Point));


    size_t numBlocks = ((numPoints) + (threadsCount - 1)) / threadsCount;
    numBlocks = (numBlocks > 65535) ? 65535 : numBlocks;
    data_kernel<<<numBlocks, threadsCount>>>(p_data, numPoints,
        leftX, upperY, rightX, lowerY);
#ifdef DEBUG
    for (int p = 0; p < numPoints; ++p)
        printf("{%f, %f}\n", p_data[p].x, p_data[p].y);
#endif
    FILE* f_ptr = fopen("points.bin", "wb");
    fwrite(p_data, sizeof(Point), numPoints, f_ptr);

    return true;
}
